#include "hip/hip_runtime.h"
#pragma once
#include ""
#include "CpuGpuMat.h"
#include "KernelGpu.cuh"
#include <math.h>


__global__ void gpuMatrixConv3D(float* image, float* mask, float* result, int imageRows, int imageCols, int maskRC, int maskDepth, int resultRows, int resultCols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float sum = 0.0;

	if (row < resultRows && col < resultCols)
	{
		int imageRowsCols = imageRows * imageCols;

		for (int maskRow = 0; maskRow < maskRC; maskRow++) {
			for (int maskCol = 0; maskCol < maskRC; maskCol++) {
				for (int dep = 0; dep < maskDepth; dep++)

					sum += image[(row + maskRow) * imageCols + col + maskCol + dep * imageRowsCols] * mask[maskRow * maskRC + maskCol + dep * maskDepth];
			}
		}
		result[row * resultCols + col] = sum;
	}
}


void gpuMatrixConvulation3D(struct CpuGpuMat* image, struct CpuGpuMat* mask, struct CpuGpuMat* result)
{
	//vscc
	int threadsPerBlock = 32;

	int gridCols = ceil(float(result->Cols) / float(threadsPerBlock));
	int gridRows = ceil(float(result->Rows) / float(threadsPerBlock));

	dim3 gridDim(gridCols, gridRows);
	dim3 blockDim(threadsPerBlock, threadsPerBlock);		// total 32*32 = 1024 threads


	gpuMatrixConv3D << < gridDim, blockDim >> > ((float*)image->gpuP, (float*)mask->gpuP, (float*)result->gpuP, image->Rows, image->Cols, mask->Rows, mask->Depth, result->Rows, result->Cols);
}